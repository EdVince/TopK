#include "hip/hip_runtime.h"

#include "topk.h"

typedef uint4 group_t; // uint32_t

void __global__ docQueryScoringCoalescedMemoryAccessSampleKernel(
        const __restrict__ uint16_t *docs, 
        const uint16_t *doc_lens, const size_t n_docs, 
        uint16_t *query, const int query_len, float *scores) {
    // each thread process one doc-query pair scoring task
    register auto tid = blockIdx.x * blockDim.x + threadIdx.x, tnum = gridDim.x * blockDim.x;

    if (tid >= n_docs) {
        return;
    }

    __shared__ uint16_t query_on_shm[MAX_QUERY_SIZE];
#pragma unroll
    for (auto i = threadIdx.x; i < query_len; i += blockDim.x) {
        query_on_shm[i] = query[i]; // not very efficient query loading temporally, as assuming its not hotspot
    }

    __syncthreads();

    for (auto doc_id = tid; doc_id < n_docs; doc_id += tnum) {
        register int query_idx = 0;

        register float tmp_score = 0.;

        register bool no_more_load = false;

        for (auto i = 0; i < MAX_DOC_SIZE / (sizeof(group_t) / sizeof(uint16_t)); i++) {
            if (no_more_load) {
                break;
            }
            register group_t loaded = ((group_t *)docs)[i * n_docs + doc_id]; // tid
            register uint16_t *doc_segment = (uint16_t*)(&loaded);
            for (auto j = 0; j < sizeof(group_t) / sizeof(uint16_t); j++) {
                if (doc_segment[j] == 0) {
                    no_more_load = true;
                    break;
                }
                while (query_idx < query_len && query_on_shm[query_idx] < doc_segment[j]) {
                    ++query_idx;
                }
                if (query_idx < query_len) {
                    tmp_score += (query_on_shm[query_idx] == doc_segment[j]);
                }
            }
            // __syncwarp();
        }
        scores[doc_id] = tmp_score / max(query_len, doc_lens[doc_id]); // tid
    }
}

void doc_query_scoring_gpu_function(std::vector<std::vector<uint16_t>> &querys,
    std::vector<std::vector<uint16_t>> &docs,
    std::vector<uint16_t> &lens,
    std::vector<std::vector<int>> &indices //shape [querys.size(), TOPK]
    ) {

    auto n_docs = docs.size();
    float* scores[2] = {nullptr,nullptr};
    float *d_scores[2] = {nullptr,nullptr};
    uint16_t *d_docs = nullptr;
    uint16_t *d_doc_lens = nullptr;
    uint16_t *d_query = nullptr;
    hipStream_t kernelStream, memcpyStream;

    // cuda第一次启动要创建context，很慢且无法避免

    std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

    // 子线程
    uint16_t *h_docs;
    std::thread convert_format([&]() {
        std::chrono::high_resolution_clock::time_point d1 = std::chrono::high_resolution_clock::now();
        h_docs = (uint16_t*)calloc(MAX_DOC_SIZE*n_docs,sizeof(uint16_t));
        #pragma omp parallel for
        for (int i = 0; i < lens.size(); i++) {
            for (int j = 0; j < lens[i]; j++) {
                auto group_sz = sizeof(group_t) / sizeof(uint16_t);
                auto layer_0_offset = j / group_sz;
                auto layer_0_stride = n_docs * group_sz;
                auto layer_1_offset = i;
                auto layer_1_stride = group_sz;
                auto layer_2_offset = j % group_sz;
                auto final_offset = layer_0_offset * layer_0_stride + layer_1_offset * layer_1_stride + layer_2_offset;
                h_docs[final_offset] = docs[i][j];
            }
        }

        hipMalloc(&d_doc_lens, sizeof(uint16_t) * n_docs);
        hipMalloc(&d_query, sizeof(uint16_t) * MAX_QUERY_SIZE);
        hipMalloc(&d_scores[0], sizeof(float) * n_docs);
        hipMalloc(&d_scores[1], sizeof(float) * n_docs);
        hipMemcpy(d_doc_lens, lens.data(), sizeof(uint16_t) * n_docs, hipMemcpyHostToDevice);

        std::chrono::high_resolution_clock::time_point d2 = std::chrono::high_resolution_clock::now();
        std::cout << "[CUDA] convert: " << std::chrono::duration_cast<std::chrono::milliseconds>(d2 - d1).count() << " ms " << std::endl;
    });

    // 主线程
    std::chrono::high_resolution_clock::time_point d1 = std::chrono::high_resolution_clock::now();
    hipHostMalloc(&scores[0], n_docs * sizeof(float));
    hipHostMalloc(&scores[1], n_docs * sizeof(float));
    hipStreamCreate(&kernelStream);
    hipStreamCreate(&memcpyStream);
    std::vector<int> s_indices(n_docs);
    hipMalloc(&d_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
    std::chrono::high_resolution_clock::time_point d2 = std::chrono::high_resolution_clock::now();
    std::cout << "[CUDA] malloc: " << std::chrono::duration_cast<std::chrono::milliseconds>(d2 - d1).count() << " ms " << std::endl;

    convert_format.join();

    // 非常耗时
    hipMemcpy(d_docs, h_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs, hipMemcpyHostToDevice);

    hipDeviceProp_t device_props;
    hipGetDeviceProperties(&device_props, 0);
    hipSetDevice(0);

    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
    uint8_t memcpyWait = 1;
    uint8_t sortWait = 2;
    bool step = false;
    std::thread sort_thread;
    // number: 0 1 2 3 4 5 6 7 8 9
    // kernel: 0 1 0 1 0 1 0 1
    // memcpy:   0 1 0 1 0 1 0 1
    // sorted:     0 1 0 1 0 1 0 1
    for(auto& query : querys) {

        // host-to-device
        const size_t query_len = query.size();
        hipMemcpy(d_query, query.data(), sizeof(uint16_t) * query_len, hipMemcpyHostToDevice);
        // launch kernel
        int block = N_THREADS_IN_ONE_BLOCK;
        int grid = (n_docs + block - 1) / block;
        docQueryScoringCoalescedMemoryAccessSampleKernel<<<grid, block, 0, kernelStream>>>(d_docs, d_doc_lens, n_docs, d_query, query_len, d_scores[step]);

        // device-to-host
        if (memcpyWait == 0)
            hipMemcpyAsync(scores[!step], d_scores[!step], sizeof(float) * n_docs, hipMemcpyDeviceToHost, memcpyStream);

        // sort scores
        if (sortWait == 0) {
            sort_thread = std::thread([&]() {
                float* cur_scores = scores[step];

                // for (int i = 0; i < n_docs; ++i) s_indices[i] = i;
                // std::partial_sort(s_indices.begin(), s_indices.begin() + TOPK, s_indices.end(),
                //                 [&cur_scores](const int& a, const int& b) {
                //                     if (cur_scores[a] != cur_scores[b])
                //                         return cur_scores[a] > cur_scores[b];  // 按照分数降序排序
                //                     return a < b;  // 如果分数相同，按索引从小到大排序
                //                 });
                // std::vector<int> s_ans(s_indices.begin(), s_indices.begin() + TOPK);
                // indices.push_back(s_ans);

                // 初始下标
                std::iota(s_indices.begin(), s_indices.end(), 0);
                // 多线程分块topk排序
                const int NUM_THREADS = 8;
                int chunk_size = s_indices.size() / NUM_THREADS;
                std::vector<std::thread> threads;
                std::vector<int> merged_results(NUM_THREADS * TOPK);
                for (int i = 0; i < NUM_THREADS; ++i) {
                    threads.emplace_back([&s_indices, &cur_scores, chunk_size, i, &merged_results]() {
                        int start = i * chunk_size;
                        int end = (i == NUM_THREADS - 1) ? s_indices.size() : start + chunk_size;
                        std::partial_sort(s_indices.begin() + start, s_indices.begin() + start + TOPK, s_indices.begin() + end,
                                        [&cur_scores](const int& a, const int& b) {
                                            if (cur_scores[a] != cur_scores[b])
                                                return cur_scores[a] > cur_scores[b];  // 按照分数降序排序
                                            return a < b;  // 如果分数相同，按索引从小到大排序
                                        });
                        std::copy(s_indices.begin() + start, s_indices.begin() + start + TOPK, merged_results.begin() + i * TOPK);
                    });
                }
                for (auto& thread : threads) {
                    thread.join();
                }
                // 合并块topk排序
                std::partial_sort(merged_results.begin(), merged_results.begin() + TOPK, merged_results.end(),
                                [&cur_scores](const int& a, const int& b) {
                                    if (cur_scores[a] != cur_scores[b])
                                        return cur_scores[a] > cur_scores[b];  // 按照分数降序排序
                                    return a < b;  // 如果分数相同，按索引从小到大排序
                                });
                std::vector<int> s_ans(merged_results.begin(), merged_results.begin() + TOPK);
                indices.push_back(s_ans);
            });
        }

        // 全部同步
        hipStreamSynchronize(kernelStream);
        hipStreamSynchronize(memcpyStream);
        if (sortWait == 0)
            sort_thread.join();

        // pipeline调度
        if (memcpyWait > 0) memcpyWait--;
        if (sortWait > 0) sortWait--;

        // 切换到另一个buffer
        step = !step;
    }
    
    hipMemcpyAsync(scores[!step], d_scores[!step], sizeof(float) * n_docs, hipMemcpyDeviceToHost, memcpyStream);
    sort_thread = std::thread([&]() {
        float* cur_scores = scores[step];
        // 初始下标
        std::iota(s_indices.begin(), s_indices.end(), 0);
        // 多线程分块topk排序
        const int NUM_THREADS = 8;
        int chunk_size = s_indices.size() / NUM_THREADS;
        std::vector<std::thread> threads;
        std::vector<int> merged_results(NUM_THREADS * TOPK);
        for (int i = 0; i < NUM_THREADS; ++i) {
            threads.emplace_back([&s_indices, &cur_scores, chunk_size, i, &merged_results]() {
                int start = i * chunk_size;
                int end = (i == NUM_THREADS - 1) ? s_indices.size() : start + chunk_size;
                std::partial_sort(s_indices.begin() + start, s_indices.begin() + start + TOPK, s_indices.begin() + end,
                                [&cur_scores](const int& a, const int& b) {
                                    if (cur_scores[a] != cur_scores[b])
                                        return cur_scores[a] > cur_scores[b];  // 按照分数降序排序
                                    return a < b;  // 如果分数相同，按索引从小到大排序
                                });
                std::copy(s_indices.begin() + start, s_indices.begin() + start + TOPK, merged_results.begin() + i * TOPK);
            });
        }
        for (auto& thread : threads) {
            thread.join();
        }
        // 合并块topk排序
        std::partial_sort(merged_results.begin(), merged_results.begin() + TOPK, merged_results.end(),
                        [&cur_scores](const int& a, const int& b) {
                            if (cur_scores[a] != cur_scores[b])
                                return cur_scores[a] > cur_scores[b];  // 按照分数降序排序
                            return a < b;  // 如果分数相同，按索引从小到大排序
                        });
        std::vector<int> s_ans(merged_results.begin(), merged_results.begin() + TOPK);
        indices.push_back(s_ans);
    });

    sort_thread.join();
    hipStreamSynchronize(memcpyStream);

    sort_thread = std::thread([&]() {
        float* cur_scores = scores[!step];
        // 初始下标
        std::iota(s_indices.begin(), s_indices.end(), 0);
        // 多线程分块topk排序
        const int NUM_THREADS = 8;
        int chunk_size = s_indices.size() / NUM_THREADS;
        std::vector<std::thread> threads;
        std::vector<int> merged_results(NUM_THREADS * TOPK);
        for (int i = 0; i < NUM_THREADS; ++i) {
            threads.emplace_back([&s_indices, &cur_scores, chunk_size, i, &merged_results]() {
                int start = i * chunk_size;
                int end = (i == NUM_THREADS - 1) ? s_indices.size() : start + chunk_size;
                std::partial_sort(s_indices.begin() + start, s_indices.begin() + start + TOPK, s_indices.begin() + end,
                                [&cur_scores](const int& a, const int& b) {
                                    if (cur_scores[a] != cur_scores[b])
                                        return cur_scores[a] > cur_scores[b];  // 按照分数降序排序
                                    return a < b;  // 如果分数相同，按索引从小到大排序
                                });
                std::copy(s_indices.begin() + start, s_indices.begin() + start + TOPK, merged_results.begin() + i * TOPK);
            });
        }
        for (auto& thread : threads) {
            thread.join();
        }
        // 合并块topk排序
        std::partial_sort(merged_results.begin(), merged_results.begin() + TOPK, merged_results.end(),
                        [&cur_scores](const int& a, const int& b) {
                            if (cur_scores[a] != cur_scores[b])
                                return cur_scores[a] > cur_scores[b];  // 按照分数降序排序
                            return a < b;  // 如果分数相同，按索引从小到大排序
                        });
        std::vector<int> s_ans(merged_results.begin(), merged_results.begin() + TOPK);
        indices.push_back(s_ans);
    });

    // deallocation
    hipFree(d_docs);
    hipFree(d_query);
    hipFree(d_scores[0]);
    hipFree(d_scores[1]);
    hipFree(d_doc_lens);
    free(h_docs);

    hipStreamDestroy(kernelStream);
    hipStreamDestroy(memcpyStream);

    sort_thread.join();

    hipHostFree(scores[0]);
    hipHostFree(scores[1]);

    std::chrono::high_resolution_clock::time_point t3 = std::chrono::high_resolution_clock::now();
    std::cout << "[CUDA] preprocess: " << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() << " ms " << std::endl;
    std::cout << "[CUDA] process: " << std::chrono::duration_cast<std::chrono::milliseconds>(t3 - t2).count() << " ms " << std::endl;
}