#include "hip/hip_runtime.h"
#include "topk.h"
#include "hipcub/hipcub.hpp"

const int32_t kThreadsNumPerBlock = 256;
const int32_t kMaxBlocksNum = 8192;
const int32_t grouptopk_size = 65536;

#define GPU_KERNEL_LOOP(i, n)                                 \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
        i += blockDim.x * gridDim.x)

inline size_t GetAlignedSize(size_t size) {
    const size_t kAlignSize = 512;
    return (size + kAlignSize - 1) / kAlignSize * kAlignSize;
}

class MultiplyFunctor final {
public:
    MultiplyFunctor(int32_t num_col) : num_col_(num_col) {}
    __host__ __device__ __forceinline__ int32_t operator()(int32_t idx) const {
        return idx * num_col_;
    }

private:
    int32_t num_col_;
};

template <typename KeyType, typename ValueType>
size_t InferTempStorageForSortPairsDescending(
        int32_t num_row,
        int32_t num_col) {
    using SegmentOffsetIter = hipcub::TransformInputIterator<
        int32_t,
        MultiplyFunctor,
        hipcub::CountingInputIterator<int32_t>>;

    hipcub::CountingInputIterator<int32_t> counting_iter(0);
    MultiplyFunctor multiply_functor(num_col);
    SegmentOffsetIter segment_offset_iter(counting_iter, multiply_functor);

    size_t temp_storage_bytes = 0;
    auto err = hipcub::DeviceSegmentedRadixSort::
        SortPairsDescending<KeyType, ValueType, SegmentOffsetIter>(
            /* d_temp_storage */ nullptr,
            /* temp_storage_bytes */ temp_storage_bytes,
            /* d_keys_in */ nullptr,
            /* d_keys_out */ nullptr,
            /* d_values_in */ nullptr,
            /* d_values_out */ nullptr,
            /* num_items */ num_row * num_col,
            /* num_segments */ num_row,
            /* d_begin_offsets */ segment_offset_iter,
            /* d_end_offsets */ segment_offset_iter + 1,
            /* begin_bit */ 0,
            /* end_bit */ sizeof(KeyType) * 8,
            /* stream */ 0);

    return temp_storage_bytes;
}

template <typename KeyType, typename ValueType>
void SortPairsDescending(
    const KeyType* keys_ptr,
    const ValueType* values_ptr,
    int32_t num_row,
    int32_t num_col,
    void* temp_storage_ptr,
    int32_t temp_storage_bytes,
    KeyType* sorted_keys_ptr,
    ValueType* sorted_values_ptr,
    hipStream_t stream) {
    size_t rt_inferred_temp_storage_bytes =
        InferTempStorageForSortPairsDescending<KeyType, ValueType>(
            num_row, num_col);

    using SegmentOffsetIter = hipcub::TransformInputIterator<
        int32_t,
        MultiplyFunctor,
        hipcub::CountingInputIterator<int32_t>>;

    hipcub::CountingInputIterator<int32_t> counting_iter(0);
    MultiplyFunctor multiply_functor(num_col);
    SegmentOffsetIter segment_offset_iter(counting_iter, multiply_functor);

    auto err = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        /* d_temp_storage */ temp_storage_ptr,
        /* temp_storage_bytes */ rt_inferred_temp_storage_bytes,
        /* d_keys_in */ keys_ptr,
        /* d_keys_out */ sorted_keys_ptr,
        /* d_values_in */ values_ptr,
        /* d_values_out */ sorted_values_ptr,
        /* num_items */ num_row * num_col,
        /* num_segments */ num_row,
        /* d_begin_offsets */ segment_offset_iter,
        /* d_end_offsets */ segment_offset_iter + 1,
        /* begin_bit */ 0,
        /* end_bit */ sizeof(KeyType) * 8,
        /* stream */ stream);
}

template <typename T>
class TmpBufferManager final {
public:
    TmpBufferManager(int32_t capacity, void* ptr, const int32_t N)
      : capacity_{capacity},
        sorted_in_elem_cnt_{N},
        indices_elem_cnt_{sorted_in_elem_cnt_},
        sorted_indices_elem_cnt_{sorted_in_elem_cnt_} {
    const int32_t sorted_in_aligned_bytes =
        GetAlignedSize(sorted_in_elem_cnt_ * sizeof(T));
    const int32_t indices_aligned_bytes =
        GetAlignedSize(indices_elem_cnt_ * sizeof(int32_t));
    const int32_t sorted_indices_aligned_bytes = indices_aligned_bytes;
    sorted_in_ptr_ = reinterpret_cast<T*>(ptr);
    indices_ptr_ = reinterpret_cast<int32_t*>(
        reinterpret_cast<char*>(sorted_in_ptr_) + sorted_in_aligned_bytes);
    sorted_indices_ptr_ = reinterpret_cast<int32_t*>(
        reinterpret_cast<char*>(indices_ptr_) + indices_aligned_bytes);
    temp_storage_ptr_ = reinterpret_cast<void*>(
        reinterpret_cast<char*>(sorted_indices_ptr_) +
        sorted_indices_aligned_bytes);
    temp_storage_bytes_ = capacity_ - sorted_in_aligned_bytes -
        indices_aligned_bytes - sorted_indices_aligned_bytes;
    }
    ~TmpBufferManager() = default;

    T* SortedInPtr() const {
        return sorted_in_ptr_;
    }
    int32_t* IndicesPtr() const {
        return indices_ptr_;
    }
    int32_t* SortedIndicesPtr() const {
        return sorted_indices_ptr_;
    }
    void* TempStoragePtr() const {
        return temp_storage_ptr_;
    }
    int32_t TempStorageBytes() const {
        return temp_storage_bytes_;
    }

private:
    int32_t capacity_;

    T* sorted_in_ptr_;
    int32_t* indices_ptr_;
    int32_t* sorted_indices_ptr_;
    void* temp_storage_ptr_;

    int32_t sorted_in_elem_cnt_;
    int32_t indices_elem_cnt_;
    int32_t sorted_indices_elem_cnt_;
    int32_t temp_storage_bytes_;
};

__global__ void InitializeIndices(
        int32_t elem_cnt,
        int32_t* indices_ptr,
        int32_t instance_size) {
    GPU_KERNEL_LOOP(i, elem_cnt) {
        indices_ptr[i] = i % instance_size;
    };
}

// ALIGNPTR
int32_t* alignPtr(int32_t* ptr, uintptr_t to) {
    uintptr_t addr = (uintptr_t)ptr;
    if (addr % to) {
        addr += to - addr % to;
    }
    return (int32_t*)addr;
}

inline int32_t BlocksNum4ThreadsNum(const int32_t n) {
    return std::min(
        (n + kThreadsNumPerBlock - 1) / kThreadsNumPerBlock,
        kMaxBlocksNum);
}

template <typename T>
void topk_launcher(
    hipStream_t stream,
    const int elem_cnt,
    const int instance_size,
    const int instance_num,
    const int top_k,
    const void* input,
    void* workspace,
    void* output_index,
    void* output_value) {
    const int32_t k = std::min(top_k, instance_size);
    const uintptr_t ALIGNMENT = 32;
    int32_t* vworkspace = alignPtr((int32_t*)workspace, ALIGNMENT);
    T* tmp_buffer = (T*)vworkspace;

    TmpBufferManager<T> buf_manager(
        static_cast<int32_t>(elem_cnt), tmp_buffer, elem_cnt);

    InitializeIndices<<<
        BlocksNum4ThreadsNum(elem_cnt),
        kThreadsNumPerBlock,
        0,
        stream>>>(elem_cnt, buf_manager.IndicesPtr(), instance_size);

    SortPairsDescending(
        (const T*)input,
        buf_manager.IndicesPtr(),
        instance_num,
        instance_size,
        buf_manager.TempStoragePtr(),
        buf_manager.TempStorageBytes(),
        buf_manager.SortedInPtr(),
        buf_manager.SortedIndicesPtr(),
        stream);

    hipDeviceSynchronize();

    hipMemcpy2D(
        (int32_t*)output_index,
        k * sizeof(int32_t),
        buf_manager.SortedIndicesPtr(),
        instance_size * sizeof(int32_t),
        k * sizeof(int32_t),
        instance_num,
        hipMemcpyDefault);

    hipMemcpy2D(
        (T*)output_value,
        k * sizeof(T),
        buf_manager.SortedInPtr(),
        instance_size * sizeof(T),
        k * sizeof(T),
        instance_num,
        hipMemcpyDefault);
}

typedef uint4 group_t; // uint32_t

template <typename T, const int len>
void __global__ docQueryScoringCoalescedMemoryAccessSampleKernelBatchN(
        const __restrict__ uint16_t *docs, const uint16_t *doc_lens, const size_t n_docs, 
        const uint16_t *query_lens, 
        uint16_t *scores, T *dict) {
    register auto tid = blockIdx.x * blockDim.x + threadIdx.x, tnum = gridDim.x * blockDim.x;
    if (tid >= n_docs)
        return;

    for (auto doc_id = tid; doc_id < n_docs; doc_id += tnum) {
        register int32_t tmp_score[len] = {0};
        register bool no_more_load = false;
        for (auto i = 0; i < MAX_DOC_SIZE / (sizeof(group_t) / sizeof(uint16_t)); i++) {
            if (no_more_load) break;
            register group_t loaded = ((group_t *)docs)[i * n_docs + doc_id]; // tid
            register uint16_t *doc_segment = (uint16_t*)(&loaded);
            for (auto j = 0; j < sizeof(group_t) / sizeof(uint16_t); j++) {
                if (doc_segment[j] == 0) {
                    no_more_load = true;
                    break;
                }
                register T flag = dict[doc_segment[j]];
                #pragma unroll
                for (auto l = 0; l < len; l++) {
                    tmp_score[l] += flag&0x1; flag >>= 1;
                }
            }
            __syncwarp();
        }
        #pragma unroll
        for (auto l = 0; l < len; l++) {
            scores[l*n_docs+doc_id] = 16384 * tmp_score[l] / max(query_lens[l], doc_lens[doc_id]);
        }
    }
}

class CUDAInit {
public:
    CUDAInit(size_t N) {
        N = ((N - 1) / grouptopk_size + 1) * grouptopk_size;
        int grouptopk_batch = N / grouptopk_size;

        hipFree(0);

        hipHostMalloc(&h_docs, sizeof(uint16_t) * MAX_DOC_SIZE * N);
        hipHostMalloc(&h_query_lens, sizeof(uint16_t) * 32);
        hipHostMalloc(&h_dict8, sizeof(uint8_t) * 50000);
        hipHostMalloc(&h_dict16, sizeof(uint16_t) * 50000);
        hipHostMalloc(&h_dict32, sizeof(uint32_t) * 50000);
        hipHostMalloc(&h_grouptopk_val, sizeof(uint16_t) * grouptopk_batch * TOPK * 16);
        hipHostMalloc(&h_grouptopk_idx, sizeof(int32_t) * grouptopk_batch * TOPK * 16);
        
        hipMalloc(&d_docs, sizeof(uint16_t) * MAX_DOC_SIZE * N);
        hipMalloc(&d_query_lens, sizeof(uint16_t) * 32);
        hipMalloc(&d_doc_lens, sizeof(uint16_t) * N);
        hipMalloc(&d_scores, sizeof(uint16_t) * N * 32);
        hipMalloc(&d_dict8, sizeof(uint8_t) * 50000);
        hipMalloc(&d_dict16, sizeof(uint16_t) * 50000);
        hipMalloc(&d_dict32, sizeof(uint32_t) * 50000);
        {
            int64_t GLOBAL_WORKSPACE_SIZE = 0;
            int elem_cnt = N * 16;
            int instance_size = grouptopk_size;
            int instance_num = elem_cnt / instance_size;
            int64_t sorted_in_aligned_bytes = GetAlignedSize(elem_cnt * sizeof(uint16_t));
            int64_t indices_aligned_bytes = GetAlignedSize(elem_cnt * sizeof(int32_t));
            int64_t sorted_indices_aligned_bytes = indices_aligned_bytes;
            int64_t temp_storage_bytes = InferTempStorageForSortPairsDescending<uint16_t, int32_t>(instance_size, instance_num);
            GLOBAL_WORKSPACE_SIZE = GetAlignedSize(sorted_in_aligned_bytes + indices_aligned_bytes + sorted_indices_aligned_bytes + temp_storage_bytes);
            hipMalloc(&d_grouptopk_workspace, GLOBAL_WORKSPACE_SIZE * sizeof(uint8_t));
        }
    }

    ~CUDAInit() {
        hipHostFree(h_docs);
        hipHostFree(h_query_lens);
        hipHostFree(h_dict8);
        hipHostFree(h_dict16);
        hipHostFree(h_dict32);
        hipHostFree(h_grouptopk_val);
        hipHostFree(h_grouptopk_idx);

        hipFree(d_docs);
        hipFree(d_query_lens);
        hipFree(d_doc_lens);
        hipFree(d_scores);
        hipFree(d_dict8);
        hipFree(d_dict16);
        hipFree(d_dict32);
        hipFree(d_grouptopk_workspace);
    }

    uint16_t* h_docs = nullptr;
    uint16_t* h_query_lens = nullptr;
    uint8_t*  h_dict8 = nullptr;
    uint16_t* h_dict16 = nullptr;
    uint32_t* h_dict32 = nullptr;
    uint16_t* h_grouptopk_val = nullptr;
    int32_t*  h_grouptopk_idx = nullptr;
    
    uint16_t* d_docs = nullptr;
    uint16_t* d_query_lens = nullptr;
    uint16_t* d_doc_lens = nullptr;
    uint16_t* d_scores = nullptr;
    uint8_t*  d_dict8 = nullptr;
    uint16_t* d_dict16 = nullptr;
    uint32_t* d_dict32 = nullptr;
    uint8_t*  d_grouptopk_workspace = nullptr;
};
CUDAInit cudaInit(8500000); // 850万

void doc_query_scoring_gpu_function(std::vector<std::vector<uint16_t>> &querys,
    std::vector<std::vector<uint16_t>> &docs,
    std::vector<uint16_t> &lens,
    std::vector<std::vector<int>> &indices //shape [querys.size(), TOPK]
    ) 
{

    int n_docs = docs.size();
    n_docs = ((n_docs - 1) / grouptopk_size + 1) * grouptopk_size;
    for(int i = 0; i < (n_docs-docs.size()); i++)
        lens.emplace_back();
    int grouptopk_batch = n_docs / grouptopk_size;

    std::chrono::high_resolution_clock::time_point h1, h2;

    int64_t cT = 0;
    h1 = std::chrono::high_resolution_clock::now();
    int numThreads = std::thread::hardware_concurrency() / 4;
    int docsSize = docs.size();
    int docsPerThread = docsSize / numThreads;
    std::vector<std::thread> threads;
    for (int q = 0; q < numThreads; q++) {
        auto start = q * docsPerThread;
        auto end = start + docsPerThread;
        if (q == numThreads-1) end = docsSize;
        threads.emplace_back([&](int s, int e){
            for (int i = s; i < e; i++) {
                for (int j = 0; j < lens[i]; j++) {
                    auto group_sz = sizeof(group_t) / sizeof(uint16_t);
                    auto layer_0_offset = j / group_sz;
                    auto layer_0_stride = n_docs * group_sz;
                    auto layer_1_offset = i;
                    auto layer_1_stride = group_sz;
                    auto layer_2_offset = j % group_sz;
                    auto final_offset = layer_0_offset * layer_0_stride + layer_1_offset * layer_1_stride + layer_2_offset;
                    cudaInit.h_docs[final_offset] = docs[i][j];
                }
            }
        },start,end);
    }
    for (auto& thread : threads) thread.join();
    h2 = std::chrono::high_resolution_clock::now();
    cT = std::chrono::duration_cast<std::chrono::milliseconds>(h2 - h1).count();

    int64_t tT = 0;
    h1 = std::chrono::high_resolution_clock::now();
    hipMemcpy(cudaInit.d_doc_lens, lens.data(), sizeof(uint16_t) * n_docs, hipMemcpyHostToDevice);
    hipMemcpy(cudaInit.d_docs, cudaInit.h_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs, hipMemcpyHostToDevice);
    h2 = std::chrono::high_resolution_clock::now();
    tT = std::chrono::duration_cast<std::chrono::milliseconds>(h2 - h1).count();

    hipDeviceProp_t device_props;
    hipGetDeviceProperties(&device_props, 0);
    hipSetDevice(0);
    int block = N_THREADS_IN_ONE_BLOCK;
    int grid = (n_docs + block - 1) / block;
    int cur_pos = 0;

    int64_t kernelT = 0, topkT = 0, start_pos = 0;
    std::chrono::high_resolution_clock::time_point tt1, tt2, tt3;

    double b32T = 0;
    h1 = std::chrono::high_resolution_clock::now();
    for (; cur_pos+32 < querys.size(); cur_pos+=32) {
        tt1 = std::chrono::high_resolution_clock::now();

        memset(cudaInit.h_dict32, 0, sizeof(uint32_t) * 50000);
        for(int i = 0; i < 32; i++) {
            for(int j = 0; j < querys[cur_pos+i].size(); j++)
                cudaInit.h_dict32[querys[cur_pos+i][j]] |= 1<<i;
            cudaInit.h_query_lens[i] = querys[cur_pos+i].size();
        }
        hipMemcpy(cudaInit.d_dict32, cudaInit.h_dict32, sizeof(uint32_t) * 50000, hipMemcpyHostToDevice);
        hipMemcpy(cudaInit.d_query_lens, cudaInit.h_query_lens, sizeof(uint16_t) * 32, hipMemcpyHostToDevice);
        docQueryScoringCoalescedMemoryAccessSampleKernelBatchN<uint32_t,32><<<grid, block>>>(
            cudaInit.d_docs, cudaInit.d_doc_lens, n_docs, 
            cudaInit.d_query_lens, 
            cudaInit.d_scores, cudaInit.d_dict32);
        hipDeviceSynchronize();

        tt2 = std::chrono::high_resolution_clock::now();

        for (int b = 0; b < 2; b++) {
            topk_launcher<uint16_t>(0,
                n_docs*16,           // elem_cnt
                grouptopk_size,     // instance_size
                grouptopk_batch*16,  // instance_num
                TOPK,               // top_k
                cudaInit.d_scores+n_docs*16*b,
                cudaInit.d_grouptopk_workspace,
                cudaInit.h_grouptopk_idx,
                cudaInit.h_grouptopk_val);
                
            for(int bb = 0; bb < 16; bb++) {
                std::vector<int> top(grouptopk_batch,0);
                std::vector<int> topk(TOPK);
                for (int i = 0; i < TOPK; i++) {
                    int idx = -1;
                    int16_t val = -1;
                    for (int j = 0; j < grouptopk_batch; j++) {
                        if (cudaInit.h_grouptopk_val[bb*TOPK*grouptopk_batch+j*TOPK+top[j]] > val) {
                            val = cudaInit.h_grouptopk_val[bb*TOPK*grouptopk_batch+j*TOPK+top[j]];
                            idx = cudaInit.h_grouptopk_idx[bb*TOPK*grouptopk_batch+j*TOPK+top[j]] + j*grouptopk_size;
                        }
                    }
                    topk[i] = idx;
                    top[idx/grouptopk_size]++;
                }
                indices.emplace_back(topk);
            }
        }

        tt3 = std::chrono::high_resolution_clock::now();
        kernelT += std::chrono::duration_cast<std::chrono::microseconds>(tt2-tt1).count();
        topkT += std::chrono::duration_cast<std::chrono::microseconds>(tt3-tt2).count();
    }

    if (cur_pos != start_pos) {
        h2 = std::chrono::high_resolution_clock::now();
        b32T = (double)std::chrono::duration_cast<std::chrono::microseconds>(h2 - h1).count() / (cur_pos-start_pos);

        kernelT /= (cur_pos-start_pos);
        topkT /= (cur_pos-start_pos);
        printf("[TIME] [B32] num:%d, kernel:%ldus, topk:%ldus\n",cur_pos-start_pos,kernelT,topkT);
    }





    double b16T = 0;
    if (cur_pos+16 < querys.size()) {
        kernelT = 0;
        topkT = 0;
        start_pos = cur_pos;
        h1 = std::chrono::high_resolution_clock::now();

        tt1 = std::chrono::high_resolution_clock::now();

        memset(cudaInit.h_dict16, 0, sizeof(uint16_t) * 50000);
        for(int i = 0; i < 16; i++) {
            for(int j = 0; j < querys[cur_pos+i].size(); j++)
                cudaInit.h_dict16[querys[cur_pos+i][j]] |= 1<<i;
            cudaInit.h_query_lens[i] = querys[cur_pos+i].size();
        }
        hipMemcpy(cudaInit.d_dict16, cudaInit.h_dict16, sizeof(uint16_t) * 50000, hipMemcpyHostToDevice);
        hipMemcpy(cudaInit.d_query_lens, cudaInit.h_query_lens, sizeof(uint16_t) * 16, hipMemcpyHostToDevice);
        docQueryScoringCoalescedMemoryAccessSampleKernelBatchN<uint16_t,16><<<grid, block>>>(
            cudaInit.d_docs, cudaInit.d_doc_lens, n_docs, 
            cudaInit.d_query_lens, 
            cudaInit.d_scores, cudaInit.d_dict16);
        hipDeviceSynchronize();

        tt2 = std::chrono::high_resolution_clock::now();

        topk_launcher<uint16_t>(0,
            n_docs*16,           // elem_cnt
            grouptopk_size,     // instance_size
            grouptopk_batch*16,  // instance_num
            TOPK,               // top_k
            cudaInit.d_scores,
            cudaInit.d_grouptopk_workspace,
            cudaInit.h_grouptopk_idx,
            cudaInit.h_grouptopk_val);
            
        for(int bb = 0; bb < 16; bb++) {
            std::vector<int> top(grouptopk_batch,0);
            std::vector<int> topk(TOPK);
            for (int i = 0; i < TOPK; i++) {
                int idx = -1;
                int16_t val = -1;
                for (int j = 0; j < grouptopk_batch; j++) {
                    if (cudaInit.h_grouptopk_val[bb*TOPK*grouptopk_batch+j*TOPK+top[j]] > val) {
                        val = cudaInit.h_grouptopk_val[bb*TOPK*grouptopk_batch+j*TOPK+top[j]];
                        idx = cudaInit.h_grouptopk_idx[bb*TOPK*grouptopk_batch+j*TOPK+top[j]] + j*grouptopk_size;
                    }
                }
                topk[i] = idx;
                top[idx/grouptopk_size]++;
            }
            indices.emplace_back(topk);
        }

        cur_pos += 16;

        tt3 = std::chrono::high_resolution_clock::now();
        kernelT += std::chrono::duration_cast<std::chrono::microseconds>(tt2-tt1).count();
        topkT += std::chrono::duration_cast<std::chrono::microseconds>(tt3-tt2).count();

        h2 = std::chrono::high_resolution_clock::now();
        b16T = (double)std::chrono::duration_cast<std::chrono::microseconds>(h2 - h1).count() / (cur_pos-start_pos);

        kernelT /= (cur_pos-start_pos);
        topkT /= (cur_pos-start_pos);
        printf("[TIME] [B16] num:%d, kernel:%ldus, topk:%ldus\n",cur_pos-start_pos,kernelT,topkT);
    }








    double b8T = 0;
    if (cur_pos+8 < querys.size()) {
        kernelT = 0;
        topkT = 0;
        start_pos = cur_pos;
        h1 = std::chrono::high_resolution_clock::now();

        tt1 = std::chrono::high_resolution_clock::now();

        memset(cudaInit.h_dict8, 0, sizeof(uint8_t) * 50000);
        for(int i = 0; i < 8; i++) {
            for(int j = 0; j < querys[cur_pos+i].size(); j++)
                cudaInit.h_dict8[querys[cur_pos+i][j]] |= 1<<i;
            cudaInit.h_query_lens[i] = querys[cur_pos+i].size();
        }
        hipMemcpy(cudaInit.d_dict8, cudaInit.h_dict8, sizeof(uint8_t) * 50000, hipMemcpyHostToDevice);
        hipMemcpy(cudaInit.d_query_lens, cudaInit.h_query_lens, sizeof(uint16_t) * 8, hipMemcpyHostToDevice);
        docQueryScoringCoalescedMemoryAccessSampleKernelBatchN<uint8_t,8><<<grid, block>>>(
            cudaInit.d_docs, cudaInit.d_doc_lens, n_docs, 
            cudaInit.d_query_lens, 
            cudaInit.d_scores, cudaInit.d_dict8);
        hipDeviceSynchronize();

        tt2 = std::chrono::high_resolution_clock::now();

        topk_launcher<uint16_t>(0,
            n_docs*8,           // elem_cnt
            grouptopk_size,     // instance_size
            grouptopk_batch*8,  // instance_num
            TOPK,               // top_k
            cudaInit.d_scores,
            cudaInit.d_grouptopk_workspace,
            cudaInit.h_grouptopk_idx,
            cudaInit.h_grouptopk_val);
            
        for(int bb = 0; bb < 8; bb++) {
            std::vector<int> top(grouptopk_batch,0);
            std::vector<int> topk(TOPK);
            for (int i = 0; i < TOPK; i++) {
                int idx = -1;
                int16_t val = -1;
                for (int j = 0; j < grouptopk_batch; j++) {
                    if (cudaInit.h_grouptopk_val[bb*TOPK*grouptopk_batch+j*TOPK+top[j]] > val) {
                        val = cudaInit.h_grouptopk_val[bb*TOPK*grouptopk_batch+j*TOPK+top[j]];
                        idx = cudaInit.h_grouptopk_idx[bb*TOPK*grouptopk_batch+j*TOPK+top[j]] + j*grouptopk_size;
                    }
                }
                topk[i] = idx;
                top[idx/grouptopk_size]++;
            }
            indices.emplace_back(topk);
        }

        cur_pos += 8;

        tt3 = std::chrono::high_resolution_clock::now();
        kernelT += std::chrono::duration_cast<std::chrono::microseconds>(tt2-tt1).count();
        topkT += std::chrono::duration_cast<std::chrono::microseconds>(tt3-tt2).count();

        h2 = std::chrono::high_resolution_clock::now();
        b8T = (double)std::chrono::duration_cast<std::chrono::microseconds>(h2 - h1).count() / (cur_pos-start_pos);

        kernelT /= (cur_pos-start_pos);
        topkT /= (cur_pos-start_pos);
        printf("[TIME] [B8] num:%d, kernel:%ldus, topk:%ldus\n",cur_pos-start_pos,kernelT,topkT);
    }





    double b1T = 0;
    h1 = std::chrono::high_resolution_clock::now();
    if (cur_pos < querys.size()) {
        kernelT = 0;
        topkT = 0;
        start_pos = cur_pos;
        h1 = std::chrono::high_resolution_clock::now();

        tt1 = std::chrono::high_resolution_clock::now();

        int remain = querys.size()-cur_pos;

        memset(cudaInit.h_dict8, 0, sizeof(uint8_t) * 50000);
        for(int i = 0; i < remain; i++) {
            for(int j = 0; j < querys[cur_pos+i].size(); j++)
                cudaInit.h_dict8[querys[cur_pos+i][j]] |= 1<<i;
            cudaInit.h_query_lens[i] = querys[cur_pos+i].size();
        }
        hipMemcpy(cudaInit.d_dict8, cudaInit.h_dict8, sizeof(uint8_t) * 50000, hipMemcpyHostToDevice);
        hipMemcpy(cudaInit.d_query_lens, cudaInit.h_query_lens, sizeof(uint16_t) * remain, hipMemcpyHostToDevice);
        docQueryScoringCoalescedMemoryAccessSampleKernelBatchN<uint8_t,8><<<grid, block>>>(
            cudaInit.d_docs, cudaInit.d_doc_lens, n_docs, 
            cudaInit.d_query_lens, 
            cudaInit.d_scores, cudaInit.d_dict8);
        hipDeviceSynchronize();

        tt2 = std::chrono::high_resolution_clock::now();

        topk_launcher<uint16_t>(0,
            n_docs*remain,           // elem_cnt
            grouptopk_size,     // instance_size
            grouptopk_batch*remain,  // instance_num
            TOPK,               // top_k
            cudaInit.d_scores,
            cudaInit.d_grouptopk_workspace,
            cudaInit.h_grouptopk_idx,
            cudaInit.h_grouptopk_val);
            
        for(int bb = 0; bb < remain; bb++) {
            std::vector<int> top(grouptopk_batch,0);
            std::vector<int> topk(TOPK);
            for (int i = 0; i < TOPK; i++) {
                int idx = -1;
                int16_t val = -1;
                for (int j = 0; j < grouptopk_batch; j++) {
                    if (cudaInit.h_grouptopk_val[bb*TOPK*grouptopk_batch+j*TOPK+top[j]] > val) {
                        val = cudaInit.h_grouptopk_val[bb*TOPK*grouptopk_batch+j*TOPK+top[j]];
                        idx = cudaInit.h_grouptopk_idx[bb*TOPK*grouptopk_batch+j*TOPK+top[j]] + j*grouptopk_size;
                    }
                }
                topk[i] = idx;
                top[idx/grouptopk_size]++;
            }
            indices.emplace_back(topk);
        }

        cur_pos += remain;

        tt3 = std::chrono::high_resolution_clock::now();
        kernelT += std::chrono::duration_cast<std::chrono::microseconds>(tt2-tt1).count();
        topkT += std::chrono::duration_cast<std::chrono::microseconds>(tt3-tt2).count();

        h2 = std::chrono::high_resolution_clock::now();
        b1T = (double)std::chrono::duration_cast<std::chrono::microseconds>(h2 - h1).count() / (cur_pos-start_pos);

        kernelT /= (cur_pos-start_pos);
        topkT /= (cur_pos-start_pos);
        printf("[TIME] [B1] num:%d, kernel:%ldus, topk:%ldus\n",cur_pos-start_pos,kernelT,topkT);

    }

    printf("[TIME] convert:%ldms, transfer:%ldms\n",cT,tT);
    printf("[TIME] Batch32:%.4lfms\n",b32T/1000);
    printf("[TIME] Batch16:%.4lfms\n",b16T/1000);
    printf("[TIME] Batch8 :%.4lfms\n",b8T/1000);
    printf("[TIME] Batch1 :%.4lfms\n",b1T/1000);
}